
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel_add(const int *a, const int *b, int *c) { *c = *a + *b; }

int main() {
  int a = 1, b = 2, c;
  int *d_a, *d_b, *d_c;

  // 1. Allocate device memory
  hipMalloc(&d_a, sizeof(int));
  hipMalloc(&d_b, sizeof(int));
  hipMalloc(&d_c, sizeof(int));

  // 2. Transfer input data to device memory
  hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

  // 3. Execute kernel
  kernel_add<<<1, 1>>>(d_a, d_b, d_c);

  // 4. Transfer output data to host memory
  hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
  printf("c: %d\n", c);
}
