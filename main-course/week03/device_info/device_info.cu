
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cuda_device_runtime_api.h>

#define CHECK_CUDA(call)                                                       \
  do {                                                                         \
    hipError_t status_ = call;                                                \
    if (status_ != hipSuccess) {                                              \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__,       \
              hipGetErrorName(status_), hipGetErrorString(status_));         \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

int main() {
  int count;
  CHECK_CUDA(hipGetDeviceCount(&count));

  printf("Number of devices: %d\n", count);
  hipDeviceProp_t props[4];
  for (int i = 0; i < count; ++i) {
    printf("\tdevice %d:\n", i);
    // TODO: get and print device properties
    hipGetDeviceProperties(&props[i], i);
    printf("\tname: %s\n", props[i].name);
    printf("\t\tmultiProcessorCount: %d\n", props[i].multiProcessorCount);
    printf("\t\tmaxThreadsPerBlock: %d\n", props[i].maxThreadsPerBlock);
    printf("\t\tsharedMemPerBlock: %ld\n", props[i].sharedMemPerBlock);
  }

  return 0;
}
