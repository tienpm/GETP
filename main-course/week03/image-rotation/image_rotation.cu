#include "hip/hip_runtime.h"
#include <cstdio>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>

#include "image_rotation.h"

#define CHECK_CUDA(call)                                                       \
  do {                                                                         \
    hipError_t status_ = call;                                                \
    if (status_ != hipSuccess) {                                              \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__,       \
              hipGetErrorName(status_), hipGetErrorString(status_));         \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

// Device(GPU) pointers
static float *input_images_gpu, *output_images_gpu;

void rotate_image_naive(float *input_images, float *output_images, int W, int H,
                        float sin_theta, float cos_theta, int num_src_images) {
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;

  // Rotate images
  for (int i = 0; i < num_src_images; i++) {
    for (int dest_x = 0; dest_x < W; dest_x++) {
      for (int dest_y = 0; dest_y < H; dest_y++) {
        float xOff = dest_x - x0;
        float yOff = dest_y - y0;
        int src_x = (int)(xOff * cos_theta + yOff * sin_theta + x0);
        int src_y = (int)(yOff * cos_theta - xOff * sin_theta + y0);
        if ((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H)) {
          output_images[i * H * W + dest_y * W + dest_x] =
              input_images[i * H * W + src_y * W + src_x];
        } else {
          output_images[i * H * W + dest_y * W + dest_x] = 0.0f;
        }
      }
    }
  }
}

__global__ void rotate_image_on_cuda(float *input_images, float *output_images,
                                     int W, int H, float sin_theta,
                                     float cos_theta, int num_src_images) {
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;

  // Rotate images
  int i = blockIdx.z * blockDim.z + threadIdx.z;
  int dest_x = blockIdx.x * blockDim.x + threadIdx.x;
  int dest_y = blockIdx.y * blockDim.y + threadIdx.y;
  float xOff = dest_x - x0;
  float yOff = dest_y - y0;
  int src_x = (int)(xOff * cos_theta + yOff * sin_theta + x0);
  int src_y = (int)(yOff * cos_theta - xOff * sin_theta + y0);
  if ((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H)) {
    output_images[i * H * W + dest_y * W + dest_x] =
        input_images[i * H * W + src_y * W + src_x];
  } else {
    output_images[i * H * W + dest_y * W + dest_x] = 0.0f;
  }
}

void rotate_image(float *input_images, float *output_images, int W, int H,
                  float sin_theta, float cos_theta, int num_src_images) {
  // Remove this line after you complete the image rotation on GPU
  // rotate_image_naive(input_images, output_images, W, H, sin_theta, cos_theta,
  //                    num_src_images);

  // (TODO) Upload input images to GPU
  CHECK_CUDA(hipMemcpy(input_images_gpu, input_images,
                        W * H * num_src_images * sizeof(float),
                        hipMemcpyHostToDevice));

  // (TODO) Launch kernel on GPU
  int bsize = 32;
  dim3 gridDim(32, 32, 32);
  dim3 blockDim((W + bsize - 1) / bsize, (H + bsize - 1) / bsize,
                num_src_images);
  rotate_image_on_cuda<<<gridDim, blockDim>>>(
      input_images_gpu, output_images_gpu, W, H, sin_theta, cos_theta,
      num_src_images);
  CHECK_CUDA(hipGetLastError());

  // (TODO) Download output images from GPU
  CHECK_CUDA(hipMemcpy(output_images, output_images_gpu,
                        W * H * num_src_images * sizeof(float),
                        hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_init(int image_width, int image_height, int num_src_images) {
  // (TODO) Allocate device memory
  CHECK_CUDA(hipMalloc(&input_images_gpu, image_width * image_height *
                                               num_src_images * sizeof(float)));
  CHECK_CUDA(
      hipMalloc(&output_images_gpu,
                 image_width * image_height * num_src_images * sizeof(float)));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_cleanup() {
  // (TODO) Free device memory
  CHECK_CUDA(hipFree(input_images_gpu));
  CHECK_CUDA(hipFree(output_images_gpu));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
