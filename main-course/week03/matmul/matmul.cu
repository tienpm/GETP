#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime_api.h>

#include "matmul.h"

#define CHECK_CUDA(call)                                                       \
  do {                                                                         \
    hipError_t status_ = call;                                                \
    if (status_ != hipSuccess) {                                              \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__,       \
              hipGetErrorName(status_), hipGetErrorString(status_));         \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;

void naive_cpu_matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  for (int i = 0; i < M; i++) {
    for (int k = 0; k < K; k++) {
      for (int j = 0; j < N; j++) {
        _C[i * N + j] += _A[i * K + k] * _B[k * N + j];
      }
    }
  }
}

__global__ void cuda_matmul(float *_A, float *_B, float *_C, int M, int N,
                            int K) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;

  if (i >= M || j >= N)
    return;
  float sum = 0;
  for (int k = 0; k < K; k++) {
    sum += _A[i * K + k] * _B[k * N + j];
  }
  _C[i * N + j] = sum;
}

void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  // Remove this line after you complete the matmul on GPU
  // naive_cpu_matmul(_A, _B, _C, M, N, K);

  // (TODO) Upload A and B matrix to GPU
  CHECK_CUDA(
      hipMemcpy(A_gpu, _A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(B_gpu, _B, N * K * sizeof(float), hipMemcpyHostToDevice));

  // (TODO) Launch kernel on a GPU
  dim3 blockDim(16, 16, 1);
  dim3 gridDim((N + 15) / 16, (M + 15) / 16);
  cuda_matmul<<<gridDim, blockDim>>>(A_gpu, B_gpu, C_gpu, M, N, K);
  CHECK_CUDA(hipGetLastError());

  // (TODO) Download C matrix from GPU
  CHECK_CUDA(
      hipMemcpy(_C, C_gpu, M * N * sizeof(float), hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_init(int M, int N, int K) {
  // (TODO) Allocate device memory
  CHECK_CUDA(hipMalloc(&A_gpu, M * K * sizeof(float)));
  CHECK_CUDA(hipMalloc(&B_gpu, N * K * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_gpu, M * N * sizeof(float)));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K) {
  // (TODO) Do any post-matmul cleanup work here.
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
