#include "hip/hip_runtime.h"
#include <cstdio>
#include <mma.h>

#include "matmul.h"

using namespace nvcuda;

#define CHECK_CUDA(call)                                                       \
  do {                                                                         \
    hipError_t status_ = call;                                                \
    if (status_ != hipSuccess) {                                              \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__,       \
              hipGetErrorName(status_), hipGetErrorString(status_));         \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

// GPU configuration.

#define WARP_SIZE 32

// MMA matrix tile dimensions.

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// GEMM configuration.

#define M_TILES 256
#define N_TILES 256
#define K_TILES 256

// #define M_GLOBAL (M * M_TILES)
// #define N_GLOBAL (N * N_TILES)
// #define K_GLOBAL (K * K_TILES)
//
#define C_LAYOUT wmma::mem_row_major

// Implementation constants.

#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#define CHUNK_K 4

#define CHUNK_LINE_BYTES (CHUNK_K * K * sizeof(half))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define GLOBAL_MEM_STRIDE N_GLOBAL

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)

#define SKEW_HALF 8
// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;

void naive_cpu_matmul(half *_A, half *_B, float *_C, int M, int N, int K) {
  for (int i = 0; i < M; i++) {
    for (int k = 0; k < K; k++) {
      for (int j = 0; j < N; j++) {
        _C[i * N + j] += (float)_A[i * K + k] * (float)_B[k * N + j];
      }
    }
  }
}

__global__ void compute_gemm(const half *A, const half *B, const float *C,
                             float *D, float alpha, float beta) {
  extern __shared__ half shmem[][CHUNK_K * K + SKEW_HALF];

  // Warp and lane identification.
  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  const unsigned int laneId = threadIdx.x % WARP_SIZE;

  // Offset in shared memory from which the B matrix is stored.
  const size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

  // This pointer is used to access the C and D matrix tiles this warp computes.
  float *shmem_warp_tile_ptr = (float *)&shmem[0][0] +
                               (warpId / 2) * SHMEM_STRIDE * K * 2 +
                               (warpId % 2) * SHMEM_OFFSET;

  // This pointer is used to stream the C and D matrices block-wide tile to and
  // from shared memory.
  float *shmem_warp_stream_ptr =
      (float *)&shmem[0][0] + warpId * SHMEM_STRIDE * K;

  // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
  // each tile computation. Technically this is not generally correct (may
  // result in a loss of precision). Zero still needs to be specially handled
  // though.
  beta /= alpha;

  // Each CTA slides along the 128 x 128 tiles from the top left corner of the
  // matrix to the right and down, and selects the next tile to compute. Once
  // there's no such tile, all warps in this CTA exit.
  for (unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
    const unsigned int block_tile_i =
        ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
    const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

    // Stop when there are no more D matrix tiles to compute in this CTA.
    if (block_tile_i >= M_TILES) {
      break;
    }

    // This warp's pointer to the C matrix data to copy memory from to shared
    // memory.
    const size_t gmem_idx =
        (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
    const float *src_gmem_warp_stream_ptr = &C[gmem_idx];

    // Stream multiple C tiles to shared memory.
#pragma unroll
    for (int i = 0; i < K; i++) {
      typedef int4 copy_t;

      *((copy_t *)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId) =
          *((copy_t *)(src_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) +
            laneId);
    }

    __syncthreads();

    // These fragments will accumulate the result of A and B matrix fragment
    // multiplications along the K_GLOBAL dimension.
    wmma::fragment<wmma::accumulator, M, N, K, float> c[WARP_COL_TILES]
                                                       [WARP_ROW_TILES];

    // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
        const float *tile_ptr =
            shmem_warp_tile_ptr + i * SHMEM_STRIDE * K + j * N;

        wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
      }
    }

    __syncthreads();

    // Scale the C matrix.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
        for (int t = 0; t < c[i][j].num_elements; t++) {
          c[i][j].x[t] *= beta;
        }
      }
    }

    // Select what warp copies what matrix to shared memory.
    // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
    const half *warp_ptr = (warpId < 4) ? (&A[block_tile_i * M * K_GLOBAL] +
                                           M * K_GLOBAL * (warpId % 4) * 2)
                                        : (&B[block_tile_j * N * K_GLOBAL] +
                                           N * K_GLOBAL * (warpId % 4) * 2);

    // Go through the global K dimension by a fixed step at a time.
#pragma unroll
    for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
      // Copy slices of the A and B matrices to shared memory.
      // The first half of the warps in the CTA copy the A matrix, the rest copy
      // the B matrix.
      size_t shmem_idx =
          warpId < (WARPS_PER_BLOCK / 2)
              ? (M * (warpId % (WARPS_PER_BLOCK / 2)) * 2)
              : (N * (warpId % (WARPS_PER_BLOCK / 2)) * 2 + shmem_idx_b_off);

      // First half of the warp copies the first row / column of the matrix,
      // the second half of the warp copies the next.
      int4 *lane_ptr = (int4 *)(warp_ptr + tile_k * K +
                                (laneId / CHUNK_COPY_LINE_LANES) * K_GLOBAL) +
                       (laneId % CHUNK_COPY_LINE_LANES);

      // Shift the second half of the warp to the next row / column in the
      // shared memory.
      shmem_idx += laneId / CHUNK_COPY_LINE_LANES;

#pragma unroll
      for (int i = 0; i < ((WARP_SIZE / 2) / CHUNK_COPY_LINES_PER_WARP) * 2;
           i++) {
        // Copy 16 bytes at once in each lane.
        *((int4 *)&shmem[shmem_idx][0] + (laneId % CHUNK_COPY_LINE_LANES)) =
            *lane_ptr;

        // Advance the global memory pointer and the shared memory index.
        lane_ptr =
            (int4 *)((half *)lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP);
        shmem_idx += CHUNK_COPY_LINES_PER_WARP;
      }

      __syncthreads();

      // Compute a grid of C matrix tiles in each warp.
#pragma unroll
      for (int k_step = 0; k_step < CHUNK_K; k_step++) {
        wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major>
            a[WARP_COL_TILES];
        wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major>
            b[WARP_ROW_TILES];

#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
          size_t shmem_idx_a = (warpId / 2) * M * 2 + (i * M);
          const half *tile_ptr = &shmem[shmem_idx_a][k_step * K];

          wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_HALF);

#pragma unroll
          for (int j = 0; j < WARP_ROW_TILES; j++) {
            if (i == 0) {
              // Load the B matrix fragment once, because it is going to be
              // reused against the other A matrix fragments.
              size_t shmem_idx_b = shmem_idx_b_off +
                                   (WARP_ROW_TILES * N) * (warpId % 2) +
                                   (j * N);
              const half *tile_ptr = &shmem[shmem_idx_b][k_step * K];

              wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_HALF);
            }

            wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
          }
        }
      }

      __syncthreads();
    }

    // Store the D fragments to shared memory.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
        // Uniform, point-wise transformations of ALL fragment elements by ALL
        // threads in the warp are well-defined even though element indices
        // within fragment storage are not defined.
        for (int t = 0; t < c[i][j].num_elements; t++)
          c[i][j].x[t] *= alpha;

        float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * K + j * N;

        wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
      }
    }

    __syncthreads();

    // Now that shared memory contains all the D tiles, stream them to global
    // memory.
    float *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
    for (int i = 0; i < K; i++) {
      *((int4 *)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
          *((int4 *)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
    }

    __syncthreads();
  }
}

// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16.
//  3) Neither A nor B are transposed.
// Note: This is a less performant version of the compute_gemm kernel. It is
// designed for
//       demonstration purposes only to show the CUDA WMMA API use without
//       relying on availability of the shared memory.
__global__ void simple_wmma_gemm(half *a, half *b, float *c, float *d, int m_ld,
                                 int n_ld, int k_ld, float alpha, float beta) {
  // Leading dimensions. Packed with no transpositions.
  int lda = m_ld;
  int ldb = k_ld;
  int ldc = n_ld;

  // Tile using a 2D grid
  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

  // Declare the fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major>
      a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major>
      b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  wmma::fill_fragment(acc_frag, 0.0f);

  // Loop over k
  for (int i = 0; i < k_ld; i += WMMA_K) {
    int aCol = i;
    int aRow = warpM * WMMA_M;

    int bCol = i;
    int bRow = warpN * WMMA_N;

    // Bounds checking
    if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
      // Load the inputs
      wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
      wmma::load_matrix_sync(b_frag, b + bCol + bRow * ldb, ldb);

      // Perform the matrix multiplication
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  // Load in the current value of c, scale it by beta, and add this our result
  // scaled by alpha
  int cCol = warpN * WMMA_N;
  int cRow = warpM * WMMA_M;

  if (cRow < m_ld && cCol < n_ld) {
    wmma::load_matrix_sync(c_frag, c + cCol + cRow * ldc, ldc,
                           wmma::mem_row_major);

    for (int i = 0; i < c_frag.num_elements; i++) {
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
    }

    // Store the output
    wmma::store_matrix_sync(d + cCol + cRow * ldc, c_frag, ldc,
                            wmma::mem_row_major);
  }
}

void matmul(half *_A, half *_B, float *_C, int M, int N, int K) {
  // Remove this line after you complete the matmul on GPU
  // naive_cpu_matmul(_A, _B, _C, M, N, K);

  // (TODO) Upload A and B matrix to GPU
  CHECK_CUDA(
      hipMemcpy(A_gpu, _A, M * K * sizeof(half), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(B_gpu, _B, K * N * sizeof(half), hipMemcpyHostToDevice));

  // (TODO) Launch kernel on a GPU
  // Define block and thread dimensions
  int M_GLOBAL(M * M_TILES);
  int N_GLOBAL(N * N_TILES);
  int K_GLOBAL(K * K_TILES);
  dim3 gridDim;
  dim3 blockDim;

  // blockDim.x must be a multple of warpSize
  // 128x4 means we have 16 warps and a block computes a 64x64 output tile
  blockDim.x = 128;
  blockDim.y = 4;

  gridDim.x =
      (M_GLOBAL + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
  gridDim.y = (N_GLOBAL + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);
  simple_wmma_gemm<<<gridDim, blockDim>>>(A, B, C, D, M_GLOBAL, N_GLOBAL,
                                          K_GLOBAL, alpha, beta);

  // (TODO) Download C matrix from GPU

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_init(int M, int N, int K) {
  // (TODO) Allocate device memory
  CHECK_CUDA(hipMalloc(&A_gpu, M * K * sizeof(half)));
  CHECK_CUDA(hipMalloc(&B_gpu, K * N * sizeof(half)));
  CHECK_CUDA(hipMalloc(&C_gpu, M * N * sizeof(float)));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(half *_A, half *_B, float *_C, int M, int N, int K) {
  // (TODO) Do any post-matmul cleanup work here.
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
