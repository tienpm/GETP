
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

#define CHECK_CUDA(call)                                                       \
  do {                                                                         \
    hipError_t status_ = call;                                                \
    if (status_ != hipSuccess) {                                              \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__,       \
              hipGetErrorName(status_), hipGetErrorString(status_));         \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

__global__ void pythagoras(int *pa, int *pb, int *pc, int *presult) {
  int a = *pa;
  int b = *pb;
  int c = *pc;

  if ((a * a + b * b) == c * c)
    *presult = 1;
  else
    *presult = 0;
}

int main(int argc, char *argv[]) {
  if (argc != 4) {
    printf("Usage: %s <num 1> <num 2> <num 3>\n", argv[0]);
    return 0;
  }

  int a = atoi(argv[1]);
  int b = atoi(argv[2]);
  int c = atoi(argv[3]);
  int result = 0;

  // TODO: 1. allocate device memory
  int *pa, *pb, *pc, *presult;
  hipMalloc(&pa, sizeof(int));
  hipMalloc(&pb, sizeof(int));
  hipMalloc(&pc, sizeof(int));
  hipMalloc(&presult, sizeof(int));

  // TODO: 2. copy data to device
  hipMemcpy(pa, &a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(pb, &b, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(pc, &c, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(presult, &result, sizeof(int), hipMemcpyHostToDevice);

  // TODO: 3. launch kernel
  pythagoras<<<1, 1>>>(pa, pb, pc, presult);

  // TODO: 4. copy result back to host

  hipMemcpy(&result, presult, sizeof(int), hipMemcpyDeviceToHost);

  if (result)
    printf("YES\n");
  else
    printf("NO\n");

  return 0;
}
